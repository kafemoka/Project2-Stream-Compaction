#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {

// TODO: __global__

__global__ void upsweep_step(int d, int *x) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);
	if (k % (int) powf(2, d + 1)) {
		return;
	}
	x[k + (int) powf(2, d + 1) - 1] += x[k + (int) powf(2, d) - 1];
}

__global__ void downsweep_step(int d, int *x) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);
	if (k % (int)powf(2, d + 1)) {
		return;
	}
	int t = x[k + (int) powf(2, d) - 1];
	x[k + (int) powf(2, d) - 1] = x[k + (int) powf(2, d + 1) - 1];
	x[k + (int) powf(2, d + 1) - 1] += t;
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
	// copy everything in idata over to the GPU
	dim3 dimBlock(n);
	dim3 dimGrid(1);
	int *dev_x;
	hipMalloc((void**)&dev_x, sizeof(int) * n);

	hipMemcpy(dev_x, idata, sizeof(int) * n, hipMemcpyHostToDevice);

    // Up Sweep
	int logn = ilog2ceil(n);
	for (int d = 0; d < logn; d++) {
		upsweep_step <<<dimGrid, dimBlock>>>(d, dev_x);
	}

	//debug: peek at the array after upsweep
	//int peek[8];
	//hipMemcpy(&peek, dev_x, sizeof(int) * 8, hipMemcpyDeviceToHost);

	// Down-Sweep
	int zero[1];
	zero[0] = 0;
	hipMemcpy(&dev_x[n - 1], zero, sizeof(int) * 1, hipMemcpyHostToDevice);
	for (int d = logn - 1; d >= 0; d--) {
		downsweep_step <<<dimGrid, dimBlock>>>(d, dev_x);
	}

	hipMemcpy(odata, dev_x, sizeof(int) * n, hipMemcpyDeviceToHost);
	hipFree(dev_x);
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {
    // TODO
    return -1;
}

}
}
