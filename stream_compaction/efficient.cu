#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {

// TODO: __global__

__global__ void upsweep_step(int d, int *x) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);
	if (k % (int) powf(2, d + 1)) {
		return;
	}
	x[k + (int) powf(2, d + 1) - 1] += x[k + (int) powf(2, d) - 1];
}

__global__ void downsweep_step(int d, int *x) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);
	if (k % (int)powf(2, d + 1)) {
		return;
	}
	int t = x[k + (int) powf(2, d) - 1];
	x[k + (int) powf(2, d) - 1] = x[k + (int) powf(2, d + 1) - 1];
	x[k + (int) powf(2, d + 1) - 1] += t;
}

__global__ void fill_by_value(int val, int *x) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);
	x[k] = val;
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {

	// copy everything in idata over to the GPU.
	// we'll need to pad the device memory with 0s to get a power of 2 array size.
	int logn = ilog2ceil(n);
	int pow2 = (int)pow(2, logn);

	dim3 dimBlock(pow2);
	dim3 dimGrid(1);
	int *dev_x;
	hipMalloc((void**)&dev_x, sizeof(int) * pow2);
	fill_by_value <<<dimGrid, dimBlock >>>(0, dev_x);

	hipMemcpy(dev_x, idata, sizeof(int) * n, hipMemcpyHostToDevice);

    // Up Sweep
	for (int d = 0; d < logn; d++) {
		upsweep_step <<<dimGrid, dimBlock>>>(d, dev_x);
	}

	//debug: peek at the array after upsweep
	//int peek[8];
	//hipMemcpy(&peek, dev_x, sizeof(int) * 8, hipMemcpyDeviceToHost);

	// Down-Sweep
	int zero[1];
	zero[0] = 0;
	hipMemcpy(&dev_x[pow2 - 1], zero, sizeof(int) * 1, hipMemcpyHostToDevice);
	for (int d = logn - 1; d >= 0; d--) {
		downsweep_step <<<dimGrid, dimBlock>>>(d, dev_x);
	}

	hipMemcpy(odata, dev_x, sizeof(int) * n, hipMemcpyDeviceToHost);
	hipFree(dev_x);
}

__global__ void temporary_array(int *x, int *temp) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);
	if (x[k] != 0) {
		temp[k] = 1;
	}
	else {
		temp[k] = 0;
	}
}

__global__ void scatter(int *x, int *trueFalse, int* scan, int *out) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);
	if (trueFalse[k]) {
		out[scan[k]] = x[k];
	}
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {
	dim3 dimBlock(n);
	dim3 dimGrid(1);
	int *dev_x;
	int *dev_tmp;
	hipMalloc((void**)&dev_x, sizeof(int) * n);
	hipMalloc((void**)&dev_tmp, sizeof(int) * n);

	// copy everything in idata over to the GPU.
	hipMemcpy(dev_x, idata, sizeof(int) * n, hipMemcpyHostToDevice);

    // Step 1: compute temporary true/false array
	temporary_array << <dimGrid, dimBlock >> >(dev_x, dev_tmp);

	// Step 2: run efficient scan on the tmp array
	// TODO: expose the CUDA relevant portions of efficient scan so we don't have to shunt around
	int *trueArray = new int[n];
	int *scanArray = new int[n];
	hipMemcpy(trueArray, dev_tmp, sizeof(int) * n, hipMemcpyDeviceToHost);
	scan(n, scanArray, trueArray);

	// Step 3: scatter
	int *dev_scatter;
	hipMalloc((void**)&dev_scatter, sizeof(int) * n);

	int *dev_scan;
	hipMalloc((void**)&dev_scan, sizeof(int) * n);
	hipMemcpy(dev_scan, scanArray, sizeof(int) * n, hipMemcpyHostToDevice);

	scatter << <dimGrid, dimBlock >> >(dev_x, dev_tmp, dev_scan, dev_scatter);

	hipMemcpy(odata, dev_scatter, sizeof(int) * n, hipMemcpyDeviceToHost);

	int return_value = scanArray[n - 1];

	delete trueArray;
	delete scanArray;
	hipFree(dev_x);
	hipFree(dev_tmp);
	hipFree(dev_scan);
	hipFree(dev_scatter);

	return return_value;
}

}
}
