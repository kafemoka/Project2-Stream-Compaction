#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
namespace Naive {

// TODO: __global__

__global__ void naive_scan_step(int d, int *x, int *x_next) {
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	int offset = powf(2, d - 1);
	if (i >= offset) {
		x_next[i] = x[i - offset] + x[i];
	}
	else {
		x_next[i] = x[i];
	}
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
    // copy everything in idata over to the GPU
	dim3 dimBlock(n);
	dim3 dimGrid(1);
	int *dev_x;
	int *dev_x_next;
	hipMalloc((void**)&dev_x, sizeof(int) * n);
	hipMalloc((void**)&dev_x_next, sizeof(int) * n);

	hipMemcpy(dev_x, idata, sizeof(int) * n, hipMemcpyHostToDevice);
	hipMemcpy(dev_x_next, dev_x, sizeof(int) * n, hipMemcpyDeviceToDevice);

	// run steps.
	// no need to pad with 0s to get a power of 2 array here,
	// this can be an "unbalanced" binary tree of ops.
	int logn = ilog2ceil(n);
	for (int d = 1; d <= logn; d++) {
		naive_scan_step <<<dimGrid, dimBlock >>>(d, dev_x, dev_x_next);
		int *temp = dev_x_next;
		dev_x_next = dev_x;
		dev_x = temp;
	}

	hipMemcpy(odata + 1, dev_x, sizeof(int) * (n - 1), hipMemcpyDeviceToHost);
	odata[0] = 0;
	
	hipFree(dev_x);
	hipFree(dev_x_next);
}

}
}
